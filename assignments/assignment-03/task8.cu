
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
__global__ void add(int *a, int *b, int *c) {
    // Position 1: To write Code here later
    int Ix, Iy, index;
    int n = 16;
    Ix = blockIdx.x * blockDim.x + threadIdx.x;
    Iy = blockIdx.y * blockDim.y + threadIdx.y;
    index = Ix * blockDim.x  * gridDim.y + Iy * blockDim.y * gridDim.y ;
    int stride = 1  ;
    for (int i = index; i < n; i+=stride)
        c[i] = a[i] + b[i];
}
int main()
{
    int *a, *b, *c, *da, *db, *dc, N=16, i, j;   
    a = (int*)malloc(sizeof(int)*N); // allocate host mem
    b = (int*)malloc(sizeof(int)*N); // and assign random
    c = (int*)malloc(sizeof(int)*N); // memory
    // Write code to initialize both a and b to 1’s.
    for (i = 0; i < N; i++) {
        a[i] = b[i] = 1;
    }
    hipMalloc((void **)&da, sizeof(int)*N);
    hipMalloc((void **)&db, sizeof(int)*N);
    hipMalloc((void **)&dc, sizeof(int)*N);
    hipMemcpy(da, a, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int)*N, hipMemcpyHostToDevice);
    dim3 dimGrid(N/8, N/8, 1);
    dim3 dimBlock(N/8, N/8, 1);
    add<<<dimGrid,dimBlock>>>(da, db, dc);
    hipMemcpy(c, dc, sizeof(int)*N, hipMemcpyDeviceToHost);
    for (j = 0; j < N/4; j++) {
        for (i = 0; i < N/4; i++) {
            printf("a[%d] + b[%d] = %d\n", j*N/4+i, j*N/4+i, c[j*N/4+i]);
        }
        printf("\n");
    }
    printf("\n");
}