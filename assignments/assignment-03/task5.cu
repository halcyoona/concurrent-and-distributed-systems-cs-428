/* task-5.cu */

#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void myHelloOnGPU(int *array){
	// Position1
	array[blockIdx.x * blockDim.x + threadIdx.x] = ( blockDim.x - threadIdx.x - 1);
	
}

int main(){
	int N = 16;
	int *cpuArray = (int*)malloc(sizeof(int)*N);
	int *gpuArray;
	hipMalloc((void **)&gpuArray, sizeof(int)*N);

	// Position 2 
	myHelloOnGPU<<<N/4, N/4>>>(gpuArray);
	hipMemcpy(cpuArray, gpuArray, sizeof(int)*N, hipMemcpyDeviceToHost);

	for(int i=0; i<N; i++){
		printf("%d ", cpuArray[i]);
	}
	printf("\n");
	return 0;
}

