/*
*  name: task-6.cu
*/


#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void myHelloOnGPU(int *array){
    // Position-1
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    array[index_y * blockDim.x * blockDim.y + index_x] = 
    11 * (( blockDim.x * gridDim.x )-(( blockIdx.x * gridDim.x - blockIdx.x*1 )+ ( blockIdx.y* gridDim.x )));
}

int main(){
    int N = 16;
    int *cpuArray = (int*)malloc(sizeof(int)*N);
    int *gpuArray;
    hipMalloc((void **)&gpuArray, sizeof(int)*N);
    // Position-2
    dim3 dimGrid(N/8, N/8, 1); 
    dim3 dimBlock(N/8, N/8, 1);
    
    myHelloOnGPU<<<dimGrid, dimBlock>>>(gpuArray);
    
    hipMemcpy(cpuArray, gpuArray, sizeof(int)*N, hipMemcpyDeviceToHost);
    for (int i = 0; i < N/4; i++){
        for (int j = 0; j < N/4; j++){
            printf("%2.2d ", cpuArray[i*N/4+j]);
        }
        printf("\n");
    }
    printf("\n");
    return 0;
}


