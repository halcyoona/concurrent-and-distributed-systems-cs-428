
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
__global__ void add(int *a, int *b, int *c) {
    // Position 1: To write Code here later
    int n = 16;
    int index = blockIdx.x * blockDim.x + threadIdx.x  ;
    int stride = gridDim.x * blockDim.x;
    for (int i = index; i < n; i+=stride)
        c[i] = a[i] + b[i];
}

int main()
{
    int *a, *b, *c, *da, *db, *dc, N=16, i;
    a = (int*)malloc(sizeof(int)*N); // allocate host mem
    b = (int*)malloc(sizeof(int)*N); // and assign random
    c = (int*)malloc(sizeof(int)*N); // memory
    // Write code to initialize both a and b to 1’s.
    for (i = 0; i < N; i++) {
        a[i] = b[i] = 1;
    }
    hipMalloc((void **)&da, sizeof(int)*N);
    hipMalloc((void **)&db, sizeof(int)*N);
    hipMalloc((void **)&dc, sizeof(int)*N);
    hipMemcpy(da, a, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int)*N, hipMemcpyHostToDevice);
    dim3 dimGrid(N/8, 1, 1);
    dim3 dimBlock(N/4, 1, 1);
    
    add<<<dimGrid,dimBlock>>>(da, db, dc);
    
    hipMemcpy(c, dc, sizeof(int)*N, hipMemcpyDeviceToHost);
    for (i = 0; i < N; i++) {
        printf("a[%d] + b[%d] = %d\n", i, i, c[i]);
    }
}